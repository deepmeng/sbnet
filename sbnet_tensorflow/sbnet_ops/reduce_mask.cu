
#include <hip/hip_runtime.h>
/*

   Sparse Blocks Network
   Copyright (c) 2017, Uber Technologies, Inc.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.

*/

#ifdef GOOGLE_CUDA

#define EIGEN_USE_GPU
#define EIGEN_USE_THREADS

#include "reduce_mask.h"
#include "zero_block_counters.cu.h"
#include "reduce_mask.cu.h"
#include "tensorflow/core/util/cuda_kernel_helper.h"
#include "cuda_helpers.h"
#include "op_utils.h"

using namespace tensorflow;
using std::cout;
using std::endl;

typedef Eigen::GpuDevice GPUDevice;

// Define the GPU implementation that launches the CUDA kernel.
template <typename T> struct ReduceMaskFunctor<GPUDevice, T> {
    void operator()(const GPUDevice& d, // Device.
        const T* mask,                  // Mask array.
        int N,                          // Batch dimension of the mask.
        int H,                          // Height of the mask.
        int W,                          // Width of the mask.
        float threshold,                // Threshold for being active.
        int bOffsH0,                    // Block padding offset height, negative.
        int bOffsW0,                    // Block padding offset width, negative.
        int bSzH,                       // Block size height.
        int bSzW,                       // Block size width.
        int bStrH,                      // Block stride, height.
        int bStrW,                      // Block stride, width.
        int bCntH,                      // Number of blocks, height.
        int bCntW,                      // Number of blocks, width.
        unsigned int numBins,           // number of bins in binCounts
        unsigned int binSize,           // maximum size of each counter bin
        short* activeBlockIndices,      // triples of [n, ih, iw] indices for active blocks.
        int* binCounts,                 // Number of indices of active blocks.
        bool avgPool                    // true for avg pooling, false for max pooling
        )
    {
        gpuErrorCheck( cudaPeekAtLastError() );

        // TODO
        // We can do better here in terms of grid/block partitioning but this is not currently a perf bottleneck
        //printf("++++++++++++++++++++++++++++++ Launching ZBC, binCounts=%x\n", binCounts);
        cudaStream_t stream = d.stream();
        gpuErrorCheck( cudaPeekAtLastError() );

        zeroBlockCounters<<<1, 32, 0, stream>>>(numBins, (unsigned int*) binCounts);
        gpuErrorCheck( cudaPeekAtLastError() );

        dim3 block(std::min(DIVUP(bSzH*bSzW, 32)*32, 1024), 1, 1);
        dim3 grid(bCntW, bCntH, N);
        reduceMask<<<grid, block, 0, d.stream()>>>(mask, N, H, W, // C is assumed to be 1
            threshold, // value to consider non-sparse block
            numBins,   // number of bins to partition activeBlockIndices to reduce atomics pressure
            binSize,
            (unsigned int*) binCounts, // counts for sub-blocks, initialized to 0
            (short*) activeBlockIndices,
            bOffsH0,
            bOffsW0,      // generally negative - first block element offset for correct padding
            bSzH, bSzW,   // block sizes
            bStrH, bStrW, // block strides
            bCntH, bCntW, // block counts
            avgPool);

        gpuErrorCheck( cudaPeekAtLastError() );
    }
};

// Instantiate functors for the types of OpKernels registered.
typedef Eigen::GpuDevice GPUDevice;
template struct ReduceMaskFunctor<GPUDevice, float>;

#endif // GOOGLE_CUDA
